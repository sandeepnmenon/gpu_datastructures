#include "hip/hip_runtime.h"
#include <memory>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

enum class probing_state
{
    SUCCESS,
    DUPLICATE,
    CONTINUE
};

template <typename T1, typename T2>
struct Pair
{
    T1 first;
    T2 second;

    __device__ Pair() : first(T1()), second(T2()) {}

    __device__ Pair(const T1 &a, const T2 &b) : first(a), second(b) {}
};

template <typename Key, typename Value>
struct Bucket
{
    Key key;
    Value value;
    // Other fields or methods for atomic operations

    __device__ Pair<Key, Value> load(std::memory_order order) const
    {
        return Pair<Key, Value>(key, value);
    }

    __device__ bool compare_exchange_strong(Pair<Key, Value> expected,
                                            Pair<Key, Value> desired,
                                            std::memory_order order)
    {
        return atomicCAS(reinterpret_cast<unsigned long long int *>(this),
                         *reinterpret_cast<unsigned long long int *>(&expected),
                         *reinterpret_cast<unsigned long long int *>(&desired)) ==
               *reinterpret_cast<unsigned long long int *>(&expected);
    }
};

template <typename T>
__host__ __device__ unsigned int hash_custom(T key)
{
    return static_cast<unsigned int>(key) * 2654435761u;
}

__device__ static constexpr int empty_sentinel = -1; // Or any other appropriate value

template <typename Key, typename Value>
class Hashmap
{
public:
    Hashmap(size_t capacity);
    ~Hashmap();

    __device__ bool insert(Key k, Value v);

    __device__ bool insert(cg::thread_block_tile<4> group, Key k, Value v);

    __device__ Value find(Key k);

    __device__ bool erase(cg::thread_block_tile<4> group, Key k);

    void getValues(const thrust::device_vector<Key> &keys, thrust::device_vector<Value> &results);

    Bucket<Key, Value> *buckets;
    size_t capacity{};

    void printAllBucketValues()
    {
        printf("Printing all %lu: bucket values\n", capacity);
        Bucket<Key, Value> *host_buckets;
        hipHostMalloc(&host_buckets, capacity * sizeof(Bucket<Key, Value>)); // Allocate pinned host memory
        hipMemcpy(host_buckets, this->buckets, capacity * sizeof(Bucket<Key, Value>), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        for (int i = 0; i < capacity; i++)
        {
            auto old_kv = host_buckets[i];
            printf("key:%d value:%d i: %d\n", old_kv.key, old_kv.value, i);
        }
        printf("Over...\n");
        hipHostFree(host_buckets); // Free the allocated host memory
    }
};

template <typename Key, typename Value>
Hashmap<Key, Value>::Hashmap(size_t cap) : capacity{cap}, buckets(nullptr)
{
    hipMalloc(&buckets, capacity * sizeof(Bucket<Key, Value>));
    hipMemset(buckets, empty_sentinel, capacity * sizeof(Bucket<Key, Value>)); // Initialize to default values
}

template <typename Key, typename Value>
Hashmap<Key, Value>::~Hashmap()
{
    hipFree(buckets);
}

template <typename Key, typename Value>
__device__ bool Hashmap<Key, Value>::insert(Key k, Value v)
{
    // get initial probing position from the hash value of the key
    auto i = hash_custom(k) % capacity;
    while (true)
    {
        // load the content of the bucket at the current probe position
        auto old_kv = buckets[i].load(std::memory_order_relaxed);
        // if the bucket is empty we can attempt to insert the pair
        if (old_kv.first == empty_sentinel)
        {
            // try to atomically replace the current content of the bucket with the input pair
            Pair<Key, Value> desired(k, v);
            bool const success = buckets[i].compare_exchange_strong(
                old_kv, desired, std::memory_order_relaxed);
            if (success)
            {
                // store was successful
                return true;
            }
        }
        else if (old_kv.first == k)
        {
            // input key is already present in the map
            return false;
        }
        // if the bucket was already occupied move to the next (linear) probing position
        // using the modulo operator to wrap back around to the beginning if we
        // go beyond the capacity
        i = ++i % capacity;
    }
}

template <typename Key, typename Value>
__device__ bool Hashmap<Key, Value>::insert(cg::thread_block_tile<4> group, Key k, Value v)
{
    // get initial probing position from the hash value of the key
    auto i = (hash_custom(k) + group.thread_rank()) % capacity;
    auto state = probing_state::CONTINUE;
    //printf("inserting key:%d value:%d i: %d\n", k, v, i);

    while (true)
    {
        // load the contents of the bucket at the current probe position of each rank in a coalesced manner
        auto old_kv = buckets[i].load(std::memory_order_relaxed);
        // input key is already present in the map
        if (group.any(old_kv.first == k))
            return false;
        // each rank checks if its current bucket is empty, i.e., a candidate bucket for insertion
        auto const empty_mask = group.ballot(old_kv.first == empty_sentinel);
        // it there is an empty buckets in the group's current probing window
        if (empty_mask)
        {
            // elect a candidate rank (here: thread with lowest rank in mask)
            auto const candidate = __ffs(empty_mask) - 1;
            //printf("candidate: %d, rank: %d key: %d value: %d i: %d\n", candidate, group.thread_rank(), k, v, i);
            if (group.thread_rank() == candidate)
            {
                // attempt atomically swapping the input Pair into the bucket
                Pair<Key, Value> desired(k, v);
                bool const success = buckets[i].compare_exchange_strong(
                    old_kv, desired, std::memory_order_relaxed);
                if (success)
                {
                    //printf("inserted key:%d value:%d i: %d\n", k, v, i);
                    // insertion went successful
                    state = probing_state::SUCCESS;
                }
                else if (old_kv.first == k)
                {
                    //printf("duplicate key:%d value:%d i: %d\n", k, v, i);
                    // else, re-check if a duplicate key has been inserted at the current probing position
                    state = probing_state::DUPLICATE;
                }
            }
            // broadcast the insertion result from the candidate rank to all other ranks
            auto const candidate_state = group.shfl(state, candidate);
            //printf("candidate_state: %d key: %d value: %d i: %d\n", candidate_state, k, v, i);
            if (candidate_state == probing_state::SUCCESS)
                return true;
            if (candidate_state == probing_state::DUPLICATE)
                return false;
        }
        else
        {
            //printf("continuing key:%d value:%d i: %d\n", k, v, i);
            // else, move to the next (linear) probing window
            i = (i + group.size()) % capacity;
        }
    }
}

template <typename Key, typename Value>
__device__ Value Hashmap<Key, Value>::find(Key k)
{
    auto i = hash_custom(k) % capacity;
    while (true)
    {
        auto old_kv = buckets[i].load(std::memory_order_relaxed);
        if (old_kv.first == k)
        {
            // Found the key, return the value
            return old_kv.second;
        }
        else if (old_kv.first == empty_sentinel)
        {
            // Key not found
            return empty_sentinel;
        }
        i = ++i % capacity;
    }
}

template <typename Key, typename Value>
__global__ void findKernel(Hashmap<Key, Value> *hashmap, const Key *keys, Value *results, int numValues)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < numValues)
    {
        results[idx] = hashmap->find(keys[idx]);
    }
}

template <typename Key, typename Value>
void Hashmap<Key, Value>::getValues(const thrust::device_vector<Key> &keys, thrust::device_vector<Value> &results)
{
    int blockSize = 256;
    int gridSize = (keys.size() + blockSize - 1) / blockSize;
    findKernel<<<gridSize, blockSize>>>(this, thrust::raw_pointer_cast(keys.data()), thrust::raw_pointer_cast(results.data()), keys.size());
}
