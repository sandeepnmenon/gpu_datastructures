#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/shuffle.h>
#include <thrust/sequence.h>

void initializeData(thrust::host_vector<int> &array, size_t numElements)
{
    thrust::sequence(array.begin(), array.end());
    thrust::default_random_engine gen;
    thrust::shuffle(array.begin(), array.end(), gen);
}

void benchmarkKernel(std::function<void()> kernelFunc, const std::string &kernelName)
{
    // Start benchmark
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch kernel
    kernelFunc();

    // End benchmark
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        // handle error
        return;
    }

    std::cout << kernelName << " time: " << milliseconds << " ms\n";

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

bool checkResults(const thrust::device_vector<int> &d_results, const thrust::host_vector<int> &h_values)
{
    thrust::host_vector<int> h_results = d_results;

    bool areEqual = thrust::equal(h_results.begin(), h_results.end(), h_values.begin());
    if (areEqual)
    {
        std::cout << "Success: d_results and h_values are the same." << std::endl;
        return true;
    }
    else
    {
        std::cout << "Error: d_results and h_values differ." << std::endl;
        // Print the first 10 values
        for (int i = 0; i < 10; i++)
            std::cout << h_results[i] << " ";
        std::cout << std::endl;
        for (int i = 0; i < 10; i++)
            std::cout << h_values[i] << " ";
        std::cout << std::endl;

        return false;
    }
}