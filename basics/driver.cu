#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <map>
#include <unistd.h>
#include <functional>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_cooperative_groups.h>
#include <thrust/sequence.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <cassert>

#include "basic_hashmap.cu"
#include "utils.cuh"

namespace cg = cooperative_groups;

struct Config {

    bool defaultInsert = false;
    bool cooperativeGroupsInsert = false;
    bool defaultSearch = false;
    size_t device = 2;
    size_t threads = 1;
    size_t blocks = 1;
    size_t numElements = 1;
    static constexpr size_t cg_size = 4;
    float load = 1.0f;

} config;

std::map<char, std::function<void(const char*)>> actions;

__global__ void testIntInsertCG(const int *keys, const int *values, const size_t numElements, Hashmap<int, int> *hashmap, size_t cg_size)
{
    int idx = (threadIdx.x + blockIdx.x * blockDim.x) / cg_size;
    if (idx < numElements)
    {
        auto group = cg::tiled_partition<4>(cg::this_thread_block());
        if (!hashmap->insert(group, keys[idx], values[idx]))
        {
            printf("Insertion failed for key[%d] %d\n", idx, keys[idx]);
        }
    }
}

__global__ void testIntInsertCG_2(const int *keys, const int *values, const size_t numElements, Hashmap<int, int> *hashmap, size_t cg_size)
{
    int threadId = (threadIdx.x + blockIdx.x * blockDim.x) / cg_size;
    int totalThreads = (gridDim.x * blockDim.x) / cg_size; // Total number of active threads

    for (int idx = threadId; idx < numElements; idx += totalThreads)
    {
        auto group = cg::tiled_partition<4>(cg::this_thread_block());
        if (!hashmap->insert(group, keys[idx], values[idx]))
        {
            printf("Insertion failed for key[%d] %d\n", idx, keys[idx]);
        }
    }
}

__global__ void testIntInsert(const int *keys, const int *values, const size_t numElements, Hashmap<int, int> *hashmap)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < numElements)
    {
        hashmap->insert(keys[idx], values[idx]);
    }
}

__global__ void testIntInsert_2(const int *keys, const int *values, const size_t numElements, Hashmap<int, int> *hashmap)
{
    int threadId = threadIdx.x + blockIdx.x * blockDim.x; // unique thread id
    int totalThreads = gridDim.x * blockDim.x; // Total number of active threads

    // Loop over elements. Each thread handles multiple insertions.
    for (int idx = threadId; idx < numElements; idx += totalThreads)
    {
        hashmap->insert(keys[idx], values[idx]);
    }
}

void insertionBenchmarkFunc(Hashmap<int, int> *hashmap, const thrust::device_vector<int> &d_keys, const thrust::device_vector<int> &d_values)
{
    // Define grid and block sizes
    int numElements = config.numElements;
    int blockSize = config.threads;
    int gridSize = (numElements + blockSize - 1) / blockSize;

    std::cout << std::setw(25) << "Threads per block:" << blockSize << "\n";
    std::cout << std::setw(25) << "Number of blocks:" << gridSize << "\n";
    std::cout << std::setw(25) << "Total number of threads:" <<  blockSize * gridSize << "\n";

    testIntInsert<<<gridSize, blockSize>>>(thrust::raw_pointer_cast(d_keys.data()), thrust::raw_pointer_cast(d_values.data()), numElements, hashmap);
    hipDeviceSynchronize();
}

void insertionBenchmarkFunc_2(Hashmap<int, int> *hashmap, const thrust::device_vector<int> &d_keys, const thrust::device_vector<int> &d_values)
{
    // Define grid and block sizes
    int numElements = config.numElements;
    int blockSize = config.threads;
    int gridSize = config.blocks;

    std::cout << std::setw(25) << "Threads per block:" << blockSize << "\n";
    std::cout << std::setw(25) << "Number of blocks:" << gridSize << "\n";
    std::cout << std::setw(25) << "Total number of threads:" <<  blockSize * gridSize << "\n";

    testIntInsert_2<<<gridSize, blockSize>>>(thrust::raw_pointer_cast(d_keys.data()), thrust::raw_pointer_cast(d_values.data()), numElements, hashmap);
    hipDeviceSynchronize();
}

void insertionBenchmarkCGFunc(Hashmap<int, int> *hashmap, const thrust::device_vector<int> &d_keys, const thrust::device_vector<int> &d_values)
{
    // Define default grid and block sizes
    int numElements = config.numElements;
    int blockSize = config.threads;
    int gridSize = (numElements * config.cg_size + blockSize - 1) / blockSize;

    assert(blockSize >= 4); // make sure there are at least 4 threads for the cooperative insert

    std::cout << std::setw(25) << "Threads per block:" << blockSize << "\n";
    std::cout << std::setw(25) << "Number of blocks:" << gridSize << "\n";
    std::cout << std::setw(25) << "Total number of threads:" <<  blockSize * gridSize << "\n";

    testIntInsertCG<<<gridSize, blockSize>>>(thrust::raw_pointer_cast(d_keys.data()), thrust::raw_pointer_cast(d_values.data()), numElements, hashmap, config.cg_size);
    hipDeviceSynchronize();
}

void insertionBenchmarkCGFunc_2(Hashmap<int, int> *hashmap, const thrust::device_vector<int> &d_keys, const thrust::device_vector<int> &d_values)
{
    // Define default grid and block sizes
    int numElements = config.numElements;
    int blockSize = config.threads;
    int gridSize = config.blocks;

    assert(blockSize >= 4); // make sure there are at least 4 threads for the cooperative insert

    std::cout << std::setw(25) << "Threads per block:" << blockSize << "\n";
    std::cout << std::setw(25) << "Number of blocks:" << gridSize << "\n";
    std::cout << std::setw(25) << "Total number of threads:" <<  blockSize * gridSize << "\n";

    testIntInsertCG_2<<<gridSize, blockSize>>>(thrust::raw_pointer_cast(d_keys.data()), thrust::raw_pointer_cast(d_values.data()), numElements, hashmap, config.cg_size);
    hipDeviceSynchronize();
}

void searchBenchMarkFunc(Hashmap<int, int> *hashmap, const thrust::device_vector<int> &d_keys, thrust::device_vector<int> &d_results)
{
    hashmap->getValues(d_keys, d_results);
    hipDeviceSynchronize();
}

void setupActions() {
    actions['d'] = [](const char*) { config.defaultInsert = true; std::cout << "Default insert\n"; };
    actions['c'] = [](const char*) { config.cooperativeGroupsInsert = true; std::cout << "Cooperative groups insert\n"; };
    actions['s'] = [](const char*) { config.defaultSearch = true; std::cout << "Default search\n"; };
    actions['n'] = [](const char* arg) { config.numElements = std::stoul(arg); };
    actions['l'] = [](const char* arg) { config.load = std::stof(arg); };
    actions['t'] = [](const char* arg) { config.threads = std::stoul(arg); };
    actions['b'] = [](const char* arg) { config.blocks = std::stoul(arg); };
    actions['g'] = [](const char* arg) { config.device = std::stoul(arg); };
}

int main(int argc, char **argv)
{   
    // Set CUDA Device - Ensure this is valid for your system
    hipError_t cudaStatus = hipSetDevice(config.device);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipSetDevice failed!" << std::endl;
        return 1;
    }

    setupActions();

    int opt;
    while ((opt = getopt(argc, argv, "dcsn:l:t:b:g:")) != -1) {
        auto action = actions.find(opt);
        if (action != actions.end())
            action->second(optarg);
        else
            std::cerr << "Unknown option: " << static_cast<char>(opt) << '\n';
    }

    // Define hashmap capacity
    std::size_t const capacity = std::ceil(config.numElements / config.load);

    std::cout << std::left; // Align text to the left
    std::cout << std::setw(25) << "Using device:" << config.device << "\n";
    std::cout << std::setw(25) << "Elements to insert:" << config.numElements << "\n";
    std::cout << std::setw(25) << "Load factor:" << config.load << "\n";
    std::cout << std::setw(25) << "Capacity:" << capacity << "\n";

    // Initialize data
    thrust::host_vector<int> h_keys(config.numElements), h_values(config.numElements);

    // Fill keys and values with test data
    initializeData(h_keys, config.numElements);
    initializeData(h_values, config.numElements);

    // Copy data from host to device
    thrust::device_vector<int> d_keys = h_keys;
    thrust::device_vector<int> d_values = h_values;

    // Create and initialize hashmap
    Hashmap<int, int> *hashmap; // Assuming constructor initializes the GPU memory
    hipMallocManaged(&hashmap, sizeof(Hashmap<int, int>));
    new (hashmap) Hashmap<int, int>(capacity);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error malloc memcpy: " << hipGetErrorString(err) << std::endl;
        // handle error
    }

    if (config.defaultInsert)
        benchmarkKernel([&]()
                        { insertionBenchmarkFunc_2(hashmap, d_keys, d_values); },
                         "non-CG Insertion");
        

    if (config.cooperativeGroupsInsert)
        benchmarkKernel([&]()
                        { std::cout << std::setw(25) << "Cooperative group size:" << config.cg_size << "\n";
                          insertionBenchmarkCGFunc_2(hashmap, d_keys, d_values); },
                        "Insertion CG");

    if (config.defaultSearch)
    {
        thrust::device_vector<int> d_results(d_keys.size());
        benchmarkKernel([&]()
                        { searchBenchMarkFunc(hashmap, d_keys, d_results); },
                        "Search");

        if (!checkResults(d_results, h_values))
        {
            // hashmap->printAllBucketValues();
        }
    }

    // Cleanup
    hashmap->~Hashmap();
    hipFree(hashmap);

    return 0;
}
