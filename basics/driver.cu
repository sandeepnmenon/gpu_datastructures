#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_cooperative_groups.h>
#include "basic_hashmap.cu"

namespace cg = cooperative_groups;

__global__ void testIntInsert(int *keys, int *values, size_t numElements, Hashmap<int, int> *hashmap)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < numElements)
    {
        auto group = cg::tiled_partition<4>(cg::this_thread_block());
        hashmap->insert(group, keys[idx], values[idx]);
    }
}


int main()
{
    // Initialize data
    const size_t numElements = 10000; // Adjust as needed
    int *keys = new int[numElements];
    int *values = new int[numElements];

    // Fill keys and values with test data
    // ...

    // Allocate memory on GPU and copy data
    int *d_keys;
    int *d_values;
    hipMalloc(&d_keys, numElements * sizeof(int));
    hipMalloc(&d_values, numElements * sizeof(int));
    hipMemcpy(d_keys, keys, numElements * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, values, numElements * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block sizes
    int blockSize = 256;
    int gridSize = (numElements + blockSize - 1) / blockSize;

    // Create and initialize hashmap
    size_t capacity = 10000; // Or any other size you prefer

    Hashmap<int, int> *hashmap = new Hashmap<int, int>(capacity); // Assuming constructor initializes the GPU memory
    // ...

    // Start benchmark
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch kernel
    testIntInsert<<<gridSize, blockSize>>>(d_keys, d_values, numElements, hashmap);

    // End benchmark
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Insertion time: " << milliseconds << " ms\n";

    // Cleanup
    hipFree(d_keys);
    hipFree(d_values);
    delete hashmap;
    delete[] keys;
    delete[] values;

    return 0;
}
