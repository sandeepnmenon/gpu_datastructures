#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_cooperative_groups.h>
#include "basic_hashmap.cu"

namespace cg = cooperative_groups;

__global__ void testIntInsertCG(int *keys, int *values, size_t numElements, Hashmap<int, int> *hashmap)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < numElements)
    {
        auto group = cg::tiled_partition<4>(cg::this_thread_block());
        hashmap->insert(group, keys[idx], values[idx]);
    }
}

__global__ void testIntInsert(int *keys, int *values, size_t numElements, Hashmap<int, int> *hashmap)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < numElements)
    {
        hashmap->insert(keys[idx], values[idx]);
    }
}

int main()
{
    // Initialize data
    const size_t numElements = 10000; // Adjust as needed
    int *keys = new int[numElements];
    int *values = new int[numElements];

    // Fill keys and values with test data
    for (size_t i = 0; i < numElements; i++)
    {
        keys[i] = i;
        values[i] = i;
    }

    // Allocate memory on GPU and copy data
    int *d_keys;
    int *d_values;
    hipMalloc(&d_keys, numElements * sizeof(int));
    hipMalloc(&d_values, numElements * sizeof(int));
    hipMemcpy(d_keys, keys, numElements * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, values, numElements * sizeof(int), hipMemcpyHostToDevice);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error malloc memcpy: " << hipGetErrorString(err) << std::endl;
        // handle error
    }

    // Define grid and block sizes
    int blockSize = 256;
    int gridSize = (numElements + blockSize - 1) / blockSize;

    // Create and initialize hashmap
    size_t capacity = 10000; // Or any other size you prefer

    Hashmap<int, int> *hashmap = new Hashmap<int, int>(capacity); // Assuming constructor initializes the GPU memory
    // ...

    // Start benchmark
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch kernel
    testIntInsert<<<gridSize, blockSize>>>(d_keys, d_values, numElements, hashmap);
    hipDeviceSynchronize(); // Wait for the kernel to finish
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error kernel: " << hipGetErrorString(err) << std::endl;
        // handle error
    }

    // End benchmark
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Insertion time: " << milliseconds << " ms\n";

    // Cleanup
    hipFree(d_keys);
    hipFree(d_values);
    delete hashmap;
    delete[] keys;
    delete[] values;

    return 0;
}
